
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cfloat>

const double voltsConstantMin = 0;
const double voltsConstantMax = 300;					//0-300V peak
const double omegaConstantMin = 2.0*M_PI*40;
const double omegaConstantMax = 2.0*M_PI*70;			//40-70Hz
const double phiConstant = 2.0*M_PI;					//0-2PI radians

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void runDEagent(const unsigned long int S, const unsigned long int G, const double F, const double R, const size_t N,
	double *bestScoreAndParameters_b, double *signalData_d, size_t signalLength, double *randomVector_d, const size_t randomLength,const size_t rate,
	int currentGen, double *bestAgent)
{
	extern __shared__ double agents_local[];
	size_t randomOffset = (gridDim.x*blockDim.x*currentGen*N+(blockIdx.x*N*gridDim.x+threadIdx.x*N))%randomLength;
	if(blockIdx.x==0&&threadIdx.x==0) printf("%lu/%lu\n",randomOffset,randomLength);

	if(blockIdx.x==0&&threadIdx.x==0)
		for (int i = 0; i < N+1; ++i) {
			agents_local[i] = bestAgent[i];
		}
	else
		for (int i = 0; i < N; ++i)
			agents_local[threadIdx.x*(N+1)+i] = randomVector_d[++randomOffset%randomLength];
	__syncthreads();
	double *child;
	child = (double *)malloc(sizeof(double)*N);
	if(currentGen!=0 && (blockIdx.x!=0||threadIdx.x!=0))
	{
		//  Reproduction
		double *parents[3];
		parents[0] = bestAgent;
		parents[1] = 0; parents[2] = 0;
		unsigned int randomParentIdx;
		for (int i = 1; i < 3; ++i)
		{
			do
			{
				randomParentIdx = floor(blockDim.x*randomVector_d[++randomOffset%randomLength]);
			} while(&agents_local[randomParentIdx*(N+1)] == parents[0] ||
				&agents_local[randomParentIdx*(N+1)] == parents[1] ||
				&agents_local[randomParentIdx*(N+1)] == parents[2]);
			parents[i] = &agents_local[randomParentIdx*(N+1)];
		}

		for (int i = 0; i < N; ++i) {
			double val = bestAgent[i];
			val += (F*(parents[1][i]-parents[2][i]));
			if(val<0.0) val=0.0;
			else if(val>1.0) val=1.0;
			child[i] = val;
		}
		//  Crossover
		unsigned int delta = floor(N*randomVector_d[++randomOffset%randomLength]);
		for (int i = 0; i < N; ++i)
			agents_local[threadIdx.x*(N+1)+i] =
				(delta != i && randomVector_d[++randomOffset%randomLength]>R)?child[i]:bestAgent[i];
	}
	double volts = voltsConstantMin+agents_local[threadIdx.x*(N+1)]*(voltsConstantMax - voltsConstantMin);
	double omega = omegaConstantMin+agents_local[threadIdx.x*(N+1)+1]*(omegaConstantMax - omegaConstantMin);
	double phi = agents_local[threadIdx.x*(N+1)+2]*phiConstant;
	double t,diff,accum=0.0;
	for(size_t pos = 0; pos < signalLength; pos++)
	{
		t=(double)pos/(double)rate;
		diff = volts*sin(
				omega*t+
				phi
			)-signalData_d[pos];
		accum += pow(diff,2);
	}
	agents_local[threadIdx.x*(N+1)+N] = accum;
	__syncthreads();	//Wait for all threads of the block to end
	//Calculate best agent
	if(threadIdx.x==0)
	{
		double *bestAgentOfBlock = &agents_local[0];
		for (int i = 1; i < blockDim.x; ++i)
			if(agents_local[i*(N+1)+N] < bestAgentOfBlock[N])
				bestAgentOfBlock = &agents_local[i*(N+1)];
		for (int i = 0; i < N+1; ++i)
			bestScoreAndParameters_b[blockIdx.x*(N+1)+i] = bestAgentOfBlock[i];
	}
	__syncthreads();	//Wait for all threads of the block to end
}
extern "C" void runDE(double *signalData, size_t signalLength, double *randomVector, const size_t randomLength,
	const unsigned long int S, const unsigned long int G, const double F, const double R, const size_t N, const double epsilon, const size_t rate)
{
	int nBlocks = (int)ceil((double)S/32);
	int nTpb = 32;
    double *signalData_d;
    HANDLE_ERROR( hipMalloc((void **)&signalData_d, signalLength*sizeof(double)) );
    HANDLE_ERROR( hipMemcpy( signalData_d, signalData, signalLength*sizeof(double), hipMemcpyHostToDevice) );
    double *randomVector_d;
    HANDLE_ERROR( hipMalloc((void **)&randomVector_d, randomLength*sizeof(double)) );
    HANDLE_ERROR( hipMemcpy( randomVector_d, randomVector, randomLength*sizeof(double), hipMemcpyHostToDevice) );
    double *bestScoreAndParameters_b;
    HANDLE_ERROR( hipMallocManaged(&bestScoreAndParameters_b,nBlocks*(N+1)*sizeof(double)) );

    //Use max threads per block
    struct hipDeviceProp_t prop;
    int cudaDevice;
    HANDLE_ERROR( hipGetDevice(&cudaDevice) );
    HANDLE_ERROR( hipGetDeviceProperties(&prop, cudaDevice) );
    printf("Device used: %s\nmaxThreadsPerBlock: %d\nmaxGridSize: %dx%dx%d\n",
    	prop.name, prop.maxThreadsPerBlock, prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);

    //S is for population size (number of agents)
    //Use one agent per thread, Use 32 agents per block, rounded up
    double *bestAgent;
    HANDLE_ERROR( hipMallocManaged(&bestAgent,(N+1)*sizeof(double)) );
    bestAgent[N]=DBL_MAX;
    for (int currentGen = 0; currentGen <= G; ++currentGen)
    {
    	if(currentGen == 28)
    		printf("Warning!\n");
    	printf("Generation: %d\n",currentGen);
		runDEagent<<< nBlocks, nTpb, nTpb*(N+1)*sizeof(double)>>>(S,G,F,R,N,bestScoreAndParameters_b,signalData_d,signalLength,randomVector_d,randomLength,rate,currentGen, bestAgent);
		hipDeviceSynchronize(); //Wait for all blocks to finish
		//Get best of them from bestScoreAndParameters (serial mode)
		for (int b = 0; b < nBlocks; ++b)
			if(bestScoreAndParameters_b[b*(N+1)+N] < bestAgent[N])
				for (int i = 0; i < N+1; ++i) bestAgent[i] = bestScoreAndParameters_b[b*(N+1)+i];
		double volts = voltsConstantMin+bestAgent[0]*(voltsConstantMax - voltsConstantMin);
		double omega = omegaConstantMin+bestAgent[1]*(omegaConstantMax - omegaConstantMin);
		double phi = bestAgent[2]*phiConstant;
		printf("Best agent in generation %d. %lf, %lf, %lf, score: %lf\n",
			currentGen, volts, omega, phi, bestAgent[N]);
    }
    HANDLE_ERROR( hipGetLastError() );

    HANDLE_ERROR( hipFree(signalData_d) );
    HANDLE_ERROR( hipFree(bestScoreAndParameters_b) );

    hipDeviceReset();
	return;
}
